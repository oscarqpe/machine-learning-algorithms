#include "hip/hip_runtime.h"
//
//  main.cpp
//  MLP Cuda
//
//  Created by Andre Valdivia on 19/04/16.
//  Copyright © 2016 Andre Valdivia. All rights reserved.
//


#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>



#define numSalidas 8
#define numEntradas 58
#define numCapas 2
#define lRate 0.3
#define numMaxNeuronas 58
#define numMaxColumnas (numMaxNeuronas + 1)
#define numData 609

#define euler exp(1)

using namespace std;

double rand0to1(){
    return rand()/double(RAND_MAX);
}

double* Data(double*_data, int fila, int columna ){
    const int pos = (columna + fila* (numEntradas + 1 ));
    return &_data[pos];
}
double* MLP(double* _MLP, int capa, int neurona, int columna){
    const int pos = (columna + (numMaxColumnas * neurona))+(capa * numMaxColumnas * numMaxNeuronas);
    return &_MLP[pos];
}

double* OutDelta(double* _out,int capa, int neurona){
    const int pos = neurona + capa*(numMaxNeuronas + 1);
    return &_out[pos];
}

void init(double* _MLP, int* numNeuronasPorCapa, int* numColumnasPorCapa, double* _out, double* _delta, double* entrada,double* real){
    
    //Init numNeuronasPorCapa
    for (int i = 0; i < numCapas - 1; i++) {
        printf("Ingrese el numero de neuronas: ");
        scanf("%d", &numNeuronasPorCapa[i]);
//        cout<< "Ingrese las neuronas en la capa "<< i << " : ";
//        cin >> numNeuronasPorCapa[i];
    }
    numNeuronasPorCapa[numCapas - 1] = numSalidas;
    
    //Init numColumnasPorCapa
    numColumnasPorCapa[0] = numEntradas + 1;
    for (int i = 1; i < numCapas; i++) {
        numColumnasPorCapa[i] = numNeuronasPorCapa[i-1] + 1;
    }
    
    //Init MLP
    int max = numCapas * numMaxNeuronas * numMaxColumnas;
    for (int i = 0; i < max; i++) {
        //        _MLP[i] = 0;
        _MLP[i] = rand0to1();
        //        _MLP[i] = 0.5;
    }
    
    //Init out y delta
    max = (numMaxNeuronas + 1) * numCapas;
    for (int i = 0; i < max; i ++) {
        _out[i] = 0;
        _delta[i] = 0;
    }
    
    for (int i = 0; i < numCapas; i++) {
        *OutDelta(_out, i, 0) = 1;
    }
    //Init entrada
    entrada[0] = 1;
    
    //Init real
    real[0] = 0;
    
}

void imprimirHeader(int* numNeuronasPorCapa, int* numColumnasPorCapa){
    //    cout<< endl;
    printf("\n numNeuronasPorCapa: \n");
    //    cout<< "numNeuronasPorCapa:"<<endl;
    for (int i = 0; i < numCapas; i++) {
        //        cout << numNeuronasPorCapa[i] << "\t";
        printf("%d\t",numNeuronasPorCapa[i]);
    }
    printf("\n\n numColumnasPorCapa\n");
    //    cout<<endl<<endl;
    
    //    cout<< "numColumnasPorCapa:"<<endl;
    for (int i = 0; i < numCapas; i++) {
        //        cout << numColumnasPorCapa[i] << "\t";
        printf("%d\t",numColumnasPorCapa[i]);
    }
    //    cout<<endl<<endl;
    printf("\n\n");
}

void imprimir_MLP(double* _MLP, int* numNeuronasPorCapa, int* numColumnasPorCapa, double* entrada, double* real, double* error, double* _out, double* _delta){
    //    cout<<endl<<"-------------- Imprimiendo matriz --------------"<<endl;
    //    cout<< "_MLP: "<<endl;
    printf("\n-----------Imprimiendo matriz -----------");
    printf("_MLP: \n");
    for (int i = 0; i < numCapas; i++) {
        //        cout<< "-- Capa "<<i<<" --"<<endl;
        printf("-- Capa %d --",i);
        for (int j = 0; j < numNeuronasPorCapa[i]; j++) {
            for (int k = 0; k < numColumnasPorCapa[i]; k++) {
                //                cout << *MLP(_MLP, i, j, k) <<"\t";
                printf("%f \t",*MLP(_MLP, i, j, k));
            }
            //            cout<< endl;
            printf("\n");
        }
        //        cout<< endl;
        printf("\n");
    }
    
    //    cout<< "---- Out ----" <<endl;
    printf("----- Out -----\n");
    for (int i = 0; i < numCapas; i++) {
        for (int j = 0; j < numNeuronasPorCapa[i] + 1; j++) {
            //            cout<< *OutDelta(_out, i, j)<< "\t";
            printf("%f \t",*OutDelta(_out, i, j));
        }
        printf("\n");
    }
    
    //    cout<<endl<<endl<< "---- Delta ----" <<endl;
    printf("\n\n----- Delta -----\n");
    for (int i = 0; i < numCapas; i++) {
        for (int j = 0; j < numNeuronasPorCapa[i] + 1; j++) {
            //            cout<< *OutDelta(_delta, i, j)<< "\t";
            printf("%f \t",*OutDelta(_delta, i, j));
        }
        //        cout<< endl;
        printf("\n");
    }
    
    //    cout<<endl<<endl<< "---- Entrada ----" <<endl;
    printf("\n\n----- Entrada -----\n");
    for (int i = 0; i < numEntradas + 1; i++) {
        //        cout<< entrada[i] << "\t";
        printf("%f\t",entrada[i]);
    }
    
    //    cout<<endl<<endl<< "---- Real ----" <<endl;
    printf("\n\n----- Real -----\n");
    for (int i = 0; i < numSalidas + 1; i++) {
        //        cout<< real[i] << "\t";
        printf("%f\t",real[i]);
    }
    
    //    cout<<endl<<endl<< "---- Error ----" <<endl;
    printf("\n\n----- Error -----\n");
    for (int i = 0; i < numSalidas + 1; i++) {
        //        cout<< error[i] << "\t";
        printf("%f\t",error[i]);
    }
}

void insertarEjemplo(double* _MLP, double* entrada, double* real){
    
    entrada[1] = 0.05;
    entrada[2] = 0.1;
    *MLP(_MLP, 0, 0, 0) = 0.35;
    *MLP(_MLP, 0, 0, 1) = 0.15;
    *MLP(_MLP, 0, 0, 2) = 0.20;
    *MLP(_MLP, 0, 1, 0) = 0.35;
    *MLP(_MLP, 0, 1, 1) = 0.25;
    *MLP(_MLP, 0, 1, 2) = 0.30;
    
    *MLP(_MLP, 1, 0, 0) = 0.6;
    *MLP(_MLP, 1, 0, 1) = 0.40;
    *MLP(_MLP, 1, 0, 2) = 0.45;
    *MLP(_MLP, 1, 1, 0) = 0.6;
    *MLP(_MLP, 1, 1, 1) = 0.50;
    *MLP(_MLP, 1, 1, 2) = 0.55;
    
    real[1] = 0.01;
    real[2] = 0.99;
    
}

double actFunct(double x){
    return 1/(1 + pow(euler, -x));
}

void forwardPropagation(double* _MLP, int* numNeuronasPorCapa, int* numColumnasPorCapa, double* _out, double* entrada){
    
    
    //Resetear outs
    for (int i = 0; i < numCapas; i++) {
        for (int j = 1; j <=numMaxNeuronas; j++) {
            *OutDelta(_out, i, j) = 0;
        }
    }
    //Forward capa 0
    for (int i = 0; i < numNeuronasPorCapa[0]; i++) {
        for (int j = 0; j < numColumnasPorCapa[0]; j++) {
            double a = entrada[j];
            double b = *MLP(_MLP, 0, i, j);
            *OutDelta(_out, 0, i + 1) +=  a * b;
        }
        *OutDelta(_out, 0, i + 1) = actFunct(*OutDelta(_out, 0, i + 1));
    }
    //Forward otras capas
    for (int i = 1; i < numCapas; i++) {
        for (int j = 0; j < numNeuronasPorCapa[i]; j++) {
            for (int k = 0; k < numColumnasPorCapa[i]; k++) {
                *OutDelta(_out, i, j+1) += *OutDelta(_out, i-1, k) * *MLP(_MLP, i, j, k);
            }
            *OutDelta(_out, i, j + 1) = actFunct(*OutDelta(_out, i, j + 1));
        }
    }
}

void backPropagation(double* _MLP, double* _out, double* _delta, int* numNeuronasPorCapa,int* numColumnasPorCapa, double* entrada, double* error,double* real){
    
    error[0] = 0;
    for (int i = 1; i <= numSalidas; i++) {
        error[i] = pow(real[i] - *OutDelta(_out, numCapas -1 , i), 2)/2 ;
        error[0] += error[i];
    }
    
    //BackPropagation
    
    //Delta 1ra capa
    for (int i = 1; i <= numNeuronasPorCapa[numCapas-1]; i++) {
        //        double outTmp = *OutDelta(_out, numCapas - 1, i);
        //        OutDelta(delta, numCapas-1, i) = (-(real[i] - outTmp))*(outTmp*(1 - outTmp)) ;
        *OutDelta(_delta, numCapas-1, i) = (-(real[i] - *OutDelta(_out, numCapas - 1, i)))*(*OutDelta(_out, numCapas - 1, i)*(1 - *OutDelta(_out, numCapas - 1, i))) ;
    }
    
    //Otros delta
    for (int i = numCapas - 2; i >= 0; i--) {
        for (int j = 1; j <= numNeuronasPorCapa[i]; j++) {
            double a = *OutDelta(_out, i, j) * (1 - *OutDelta(_out, i, j));//
            double b = 0;
            for (int k = 0; k < numColumnasPorCapa[i+1]; k++) {
                //                cout<< "K: " << k << "   num: "<< numColumnasPorCapa[i+1]<<endl;
                //                cout<< *MLP(_MLP, i+1, k, j) << " * " << *OutDelta(_delta, i+1, k+1) << endl;
                b += *MLP(_MLP, i+1, k, j) * *OutDelta(_delta, i+1, k+1);
            }
            *OutDelta(_delta, i, j) = a * b;
        }
    }
    
    //Actalizar primera capa de pesos
    for (int j = 0; j < numNeuronasPorCapa[0]; j++) {
        for (int k = 0; k < numColumnasPorCapa[0]; k++) {
            *MLP(_MLP, 0, j, k) = *MLP(_MLP, 0, j, k) - lRate * entrada[k] * *OutDelta(_delta, 0, j + 1);
        }
    }
    
    //Actualizar otras capas de pesos
    for (int i = 1; i < numCapas; i++) {
        for (int j = 0; j < numNeuronasPorCapa[0]; j++) {
            for (int k = 0; k < numColumnasPorCapa[0]; k++) {
                *MLP(_MLP, i, j, k) = *MLP(_MLP, i, j, k) - lRate * *OutDelta(_out, i-1, k) * *OutDelta(_delta, i, j + 1);
            }
        }
    }
}

void Train(double* _MLP,int* numNeuronasPorCapa, int* numColumnasPorCapa, double* _out,double* entrada, double* _delta,double* error, double* real){
    
    forwardPropagation(_MLP, numNeuronasPorCapa, numColumnasPorCapa, _out, entrada);
    backPropagation(_MLP, _out, _delta, numNeuronasPorCapa, numColumnasPorCapa, entrada, error, real);
}

int Test(double* _MLP,int* numNeuronasPorCapa, int* numColumnasPorCapa, double* _out,double* entrada, double* real, int posRespuesta){
    
    forwardPropagation(_MLP, numNeuronasPorCapa, numColumnasPorCapa, _out, entrada);
    double max = 0;
    int posMax = 0;
    for (int i = 1; i <= numSalidas; i++) {
        if (*OutDelta(_out, numCapas-1, i) > max) {
            max = *OutDelta(_out, numCapas-1, i);
            posMax = i;
        }
    }
    return posMax;
}

bool readDataIris(double* _data,int numTrain,int* TrainNumbers,int*TestNumbers){
    char buffer[4096] ;
    int ii=0,jj=0;
    // FILE *fstream = fopen("iris.txt","r");
    /Users/Andre/Dropbox/Citec/Circulo Investigacion/BasesDeDatos/cara.csv
    FILE *fstream = fopen("/Users/Andre/Dropbox/Citec/Circulo Investigacion/BasesDeDatos/iris.txt","r");
    if(fstream == NULL)
    {
        printf("\n file training opening failed ");
        return false;
    }
    while (fgets(buffer, 4096, fstream)){
        char* tmp = strdup(buffer);
        const char* tok;
        jj = 0;
        for (tok = strtok(tmp, ","); tok && *tok; tok = strtok(NULL, ",\n")){
            double i = atof(tok);
            if (i == 0) {
                if (strncmp (tok,"Iris-setosa",9) == 0) {
                    *Data(_data, ii, jj) = 1;
                }else if (strncmp (tok,"Iris-versicolor",9) == 0) {
                    *Data(_data, ii, jj) = 2;
                }else if (strncmp (tok,"Iris-virginica",9) == 0) {
                    *Data(_data, ii, jj) = 3;
                }
            }else{
                *Data(_data, ii, jj) = atof(tok);
            }
            jj++;
        }
        free(tmp);
        ii++;
    }
    
    bool seleccionados[numData];
    for (int i = 0; i < numData; i++) {
        seleccionados[i] = false;
    }
    int count = 0;
    
    while (true) {
        int s = rand () % numData;//Entre M y N
        if (seleccionados[s] == false){
            count++;
            seleccionados[s] = true;
        }
        if (count >= numTrain) {
            break;
        }
    }
    for (int i = 0, j = 0, k = 0; i < numData; i++) {
        if (seleccionados[i] == true) {
            TrainNumbers[j] = i;
            j++;
        }else{
            TestNumbers[k] = i;
            k++;
        }
    }
    return true  ;
}

bool readDataCara(double* _data,int numTrain,int* TrainNumbers,int*TestNumbers){
    char buffer[4096] ;
    int ii=0,jj=0;
    
    FILE *fstream = fopen("/Users/Andre/Dropbox/Citec/Circulo Investigacion/BasesDeDatos/cara.csv","r");
    // FILE *fstream = fopen("cara.csv","r");
    if(fstream == NULL)
    {
        printf("\n file training opening failed ");
        return false;
    }
    char* tmp2 = strdup(buffer);
    while (fgets(buffer, 4096, fstream)){
        char* tmp = strdup(buffer);
        const char* tok;
        jj = 0;
        for (tok = strtok(tmp, ";"); tok && *tok; tok = strtok(NULL, ";\n" )){
            double i = atof(tok);
            if (i == 0) {
                //                cout<<ii<<"\t"<<jj<<"\t Hay string: "<<tok<<endl;//Borrar
                *Data(_data, ii, jj) = atof(tok);
            }else{
                *Data(_data, ii, jj) = atof(tok);
            }
            jj++;
        }
        free(tmp);
        ii++;
    }
    free(tmp2);
    
    bool seleccionados[numData];
    for (int i = 0; i < numData; i++) {
        seleccionados[i] = false;
    }
    int count = 0;
    
    while (true) {
        int s = rand () % numData;//Entre M y N
        if (seleccionados[s] == false){
            count++;
            seleccionados[s] = true;
        }
        if (count >= numTrain) {
            break;
        }
    }
    for (int i = 0, j = 0, k = 0; i < numData; i++) {
        if (seleccionados[i] == true) {
            TrainNumbers[j] = i;
            j++;
        }else{
            TestNumbers[k] = i;
            k++;
        }
    }
    return true  ;
}

int main(int argc, const char * argv[]) {
    cout.precision(4);
    srand(time(NULL));
    double _MLP[numCapas * numMaxNeuronas * numMaxColumnas];
    double real[numSalidas + 1];
    double error[numSalidas + 1];
    double entrada[numEntradas + 1];
    double _out[(numMaxNeuronas + 1) * numCapas];
    double _delta[(numMaxNeuronas + 1) * numCapas];
    
    
    double* _data = (double*)malloc(sizeof(double)*numData*(numEntradas+1));
    //    double data[numData][numEntradas + 3];
    
    int numNeuronasPorCapa[numCapas];
    int numColumnasPorCapa[numCapas];
    init(_MLP, numNeuronasPorCapa, numColumnasPorCapa, _out, _delta, entrada,real);
    imprimirHeader(numNeuronasPorCapa,numColumnasPorCapa);
    
    //Leer data
    const int numTrain = numData/2; //Cambiar aqui para el porcentaje de entrenamiento
    const int numTest = numData - numTrain;
    int TrainNumbers[numTrain];
    int TestNumbers[numTest];
    
    
    //    if(readDataIris(_data, numTrain, TrainNumbers, TestNumbers) == false){
    //        return -1;
    //    }
    
    if(readDataCara(_data, numTrain, TrainNumbers, TestNumbers) == false){
        return -1;
    }
    
    //    cout<< "------ Imprimiendo Data -----------"<<endl;
    //    for (int i = 0; i < numData; i++) {
    //        for (int j = 0; j < numEntradas; j++) {
    //            cout<< *Data(_data, i, j) << "\t";
    //        }
    //        cout<<endl;
    //    }
    //    cout<<endl<<endl;
    
    //Entrenar
    int posClase = 0;
    
    for (int i = 0; i < 10000; i++) {
        for(int j = 0;j < numTrain;j++){
            for (int k = 1; k <= numEntradas; k++) {
                entrada[k] = *Data(_data, TrainNumbers[j], k);
            }
            int respuesta = *Data(_data, TrainNumbers[j], posClase);
            for (int k = 1; k <= numSalidas; k++) {
                real[k] = 0.01;
            }
            real[respuesta] = 0.99;
            Train(_MLP, numNeuronasPorCapa, numColumnasPorCapa, _out, entrada, _delta, error, real);
            //            imprimir_MLP(_MLP, numNeuronasPorCapa, numColumnasPorCapa, entrada, real, error, _out, _delta);
        }
    }
    
    //Testear
    int correctas = 0;
    int incorrectas = 0;
    int incorrectasPorDato[numSalidas + 1];
    for (int i = 1; i<=numSalidas; i++) {
        incorrectasPorDato[i] = 0;
    }
    for(int j = 0;j < numTest;j++){
        for (int k = 1; k <= numEntradas; k++) {
            entrada[k] = *Data(_data, TestNumbers[j], k);
        }
        int respuesta = *Data(_data, TestNumbers[j], posClase);
        for (int k = 1; k <= numSalidas; k++) {
            real[k] = 0.01;
        }
        real[respuesta] = 0.99;
        int resp = Test(_MLP, numNeuronasPorCapa, numColumnasPorCapa, _out, entrada, real, respuesta);
        //        cout<<entrada[0]<<"\t"<<entrada[1]<<"\t"<<entrada[2]<<"\t"<<entrada[3]<<"\t"<<entrada[4]<<endl;
        //        cout<<real[0]<<"\t"<<real[1]<<"\t"<<real[2]<<"\t"<<real[3]<<"\t"<<real[4]<<endl;
        //        cout<< data[TestNumbers[j]][0]<< "\t" << data[TestNumbers[j]][1]<< "\t"<< data[TestNumbers[j]][2] << "\t"<< data[TestNumbers[j]][3]<< "\t"<< data[TestNumbers[j]][4]<<endl;
        //        cout<<"Resp: "<< resp <<endl;
        //        cout<<"Respuesta real: "<< respuesta <<endl<<endl;
        if (resp == respuesta) {
            correctas++;
        }else{
            incorrectasPorDato[respuesta]++;
            incorrectas++;
        }
        //            imprimir_MLP(_MLP, numNeuronasPorCapa, numColumnasPorCapa, entrada, real, error, _out, _delta);
    }
    cout<<endl<<endl;
    cout<<"Correctas: "<< correctas<<endl;
    cout<<"Incorrectas: "<< incorrectas<<endl;
    
    for (int i = 1; i<= numSalidas; i++) {
        cout<< i<<": "<< incorrectasPorDato[i]<<endl;
    }
    double porcentaje = (double)correctas/(double)numTest * 100;
    cout<< "Un porcentaje de: "<< porcentaje <<"%"<<endl;
    //    insertarEjemplo(_MLP,entrada,real);    //Funcion insertar data ejemplo
    //    forwardPropagation(_MLP, numNeuronasPorCapa, numColumnasPorCapa, _out, entrada);
    //    backPropagation(_MLP, _out, _delta, numNeuronasPorCapa, numColumnasPorCapa, entrada, error, real);
    
    //    imprimir_MLP(_MLP, numNeuronasPorCapa, numColumnasPorCapa, entrada, real, error, _out, _delta);
    free(_MLP);
    free( _out);
    free(_delta);
    free(_data);
}
